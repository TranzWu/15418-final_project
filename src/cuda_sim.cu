#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <thrust/scan.h>
#include <thrust/device_ptr.h>

#include "cuda_sim.h"
#include "cycleTimer.h"

struct GlobalConstants {
    size_t numberOfParticles;

    float boxSize;

    float dudr;
    float r_cut;
    float u_cut;
    float delta;

    float* positions;
    float* velocities;
    float* forces;

    float* kinetic;
    float* potential;

    short* neighbors;
    short* counts;
};
__constant__ GlobalConstants params;

/* Device functions */

__device__ float calculateDistance(size_t i, size_t j)
{
    float boxSize = params.boxSize;

    float distance = 0;
    for (int k = 0; k < 3; k++){
        float diff = params.positions[i * 3 + k] - params.positions[j * 3 + k];
        if (diff < -boxSize/2) diff += boxSize;
        if (diff > boxSize/2) diff -= boxSize;
        distance += diff * diff;
    }

    distance = std::sqrt(distance);

    return distance;
}

/* Kernels */

__global__ void markNeighbors()
{
    int numberOfParticles = params.numberOfParticles;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    float r_cut = params.r_cut;
    short* neighbors = params.neighbors;
    int i = index / numberOfParticles;
    int j = index % numberOfParticles;

    float dist = calculateDistance(i, j);

    if (dist < r_cut * 2)
        neighbors[i * numberOfParticles + j] = 1;
    else
        neighbors[i * numberOfParticles + j] = 0;
}

__global__ void getNeighborCounts()
{
    short* neighbors = params.neighbors;
    short* counts = params.counts;
    int numberOfParticles = params.numberOfParticles;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= numberOfParticles) return;

    counts[i] = neighbors[i * numberOfParticles + numberOfParticles - 1];
}

__global__ void reduceNeighbors()
{
    int numberOfParticles = params.numberOfParticles;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    short* neighbors = params.neighbors;
    int i = index / numberOfParticles;
    int j = index % numberOfParticles;

    if (j == numberOfParticles - 1)
        return;

    if (neighbors[i * numberOfParticles + j] == neighbors[i * numberOfParticles + j + 1] + 1)
    {
        index = neighbors[i * numberOfParticles + j];
        neighbors[i * numberOfParticles + index] = j;
    }
}

__global__ void calculateForceKernel()
{
    int numberOfParticles = params.numberOfParticles;
    float boxSize = params.boxSize;
    float dudr = params.dudr;
    float r_cut = params.r_cut;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int i = index / (numberOfParticles / 5);
    int h = index % (numberOfParticles / 5);

    while (h < params.counts[i])
    {
        int j = params.neighbors[i * numberOfParticles + h];

        if (i >= numberOfParticles || j >= numberOfParticles) return;
        if (j <= i) return;
        if ((i % 3 == 0) && (j - i == 1) || (i % 3 == 0) && (j - i == 2) ) return;

        float dist = calculateDistance(i, j);
        if (dist >= r_cut) return;

        float u_actual = 4 * (1/std::pow(dist, 12) - 1/std::pow(dist, 6));
        // atomicAdd(params.potential, u_actual - u_cut - (dist - r_cut) * dudr);

        for (int k = 0; k < 3; k++)
        {
            float r = params.positions[i * 3 + k] - params.positions[j * 3 + k];
            if (r < -boxSize/2) r += boxSize;
            if (r > boxSize/2) r -= boxSize;

            float f = r * (48/pow(dist, 14) - 24/pow(dist, 8) + dudr/dist);
            atomicAdd(&params.forces[i * 3 + k], f);
            atomicAdd(&params.forces[j * 3 + k], -1 * f);
        }

        h += numberOfParticles / 5;
    }

}

__global__ void calculateBondAngleKernel()
{
    int numberOfParticles = params.numberOfParticles;
    float boxSize = params.boxSize;
    double spring = 1000;
    double x0 = 0.8;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= numberOfParticles * 2) return;
    int h = index % 2 + 1;
    int i = (index / 2) * 3;

    float dist = calculateDistance(i, i + h);

    for (int k = 0; k < 3; k++){
        double r = params.positions[i * 3 + k] - params.positions[(i+h) * 3 + k];
        if (r < -boxSize/2) r += boxSize;
        if (r > boxSize/2) r -= boxSize;
        atomicAdd(params.potential, 0.5 * spring * pow(dist - x0, 2));
        float f = -spring * (dist - x0) * r / dist;
        atomicAdd(&params.forces[i * 3 + k], f);
        atomicAdd(&params.forces[(i + h) * 3 + k], -f);
    }
}

__global__ void calculateKineticKernel()
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index > params.numberOfParticles * 3) return;
    int i = index / 3;
    int k = index % 3;

    atomicAdd(params.kinetic, 0.5 * pow(params.velocities[i * 3 + k], 2));
}

__global__ void updatePositionKernel()
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index > params.numberOfParticles * 3) return;
    int i = index / 3;
    int k = index % 3;

    params.positions[i * 3 + k] += params.velocities[i * 3 + k] * params.delta;
    if (params.positions[i * 3 + k] > params.boxSize){
        params.positions[i * 3 + k] -= params.boxSize;
    }
    if (params.positions[i * 3 + k] < 0){
        params.positions[i * 3 + k] += params.boxSize;
    }
}

__global__ void updateVelocityKernel()
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index > params.numberOfParticles * 3) return;
    int i = index / 3;
    int k = index % 3;

    float a = params.forces[i * 3 + k] * params.delta / 2;
    params.velocities[i * 3 + k] += a;
}

/* Interface */

void CudaSim::getPositions(float *out)
{
    hipMemcpy(out, positions, 3 * numberOfParticles * sizeof(float), hipMemcpyDeviceToHost);
}

void CudaSim::getVelocities(float *out)
{
    hipMemcpy(out, velocities, 3 * numberOfParticles * sizeof(float), hipMemcpyDeviceToHost);
}

float CudaSim::getKinetic()
{
    float out;
    hipMemcpy(&out, kinetic, sizeof(float), hipMemcpyDeviceToHost);
    return out;
}

float CudaSim::getPotential()
{
    float out;
    hipMemcpy(&out, potential, sizeof(float), hipMemcpyDeviceToHost);
    return out;
}

void printDeviceInfo()
{
    int deviceCount = 0;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}

CudaSim::CudaSim(size_t numberOfParticles_in, float boxSize_in, float dudr_in,
                 float r_cut_in, float u_cut_in, float delta_in, float* positions_init)
{
    printDeviceInfo();

    // Initialize instance variables
    numberOfParticles = numberOfParticles_in;
    boxSize = boxSize_in;
    dudr = dudr_in;
    r_cut = r_cut_in;
    u_cut = u_cut_in;
    delta = delta_in;
    hipMalloc(&positions, sizeof(float) * 3 * numberOfParticles);
    hipMemcpy(positions, positions_init, sizeof(float) * 3 * numberOfParticles, hipMemcpyHostToDevice);
    hipMalloc(&velocities, sizeof(float) * 3 * numberOfParticles);
    hipMemset(velocities, 0, sizeof(float) * 3 * numberOfParticles);
    hipMalloc(&forces, sizeof(float) * 3 * numberOfParticles);
    hipMemset(forces, 0, sizeof(float) * 3 * numberOfParticles);
    hipMalloc(&neighbors, sizeof(short) * numberOfParticles * numberOfParticles);
    hipMemset(neighbors, 0, sizeof(short) * numberOfParticles * numberOfParticles);
    hipMalloc(&counts, sizeof(short) * numberOfParticles);
    hipMemset(counts, 0, sizeof(short) * numberOfParticles);
    hipMalloc(&kinetic, sizeof(float));
    hipMemset(kinetic, 0, sizeof(float));
    hipMalloc(&potential, sizeof(float));
    hipMemset(potential, 0, sizeof(float));

    // Initialize global constants
    GlobalConstants local_params;
    local_params.boxSize = boxSize;
    local_params.dudr = dudr;
    local_params.r_cut = r_cut;
    local_params.u_cut = u_cut;
    local_params.delta = delta;
    local_params.numberOfParticles = numberOfParticles;
    local_params.positions = positions;
    local_params.velocities = velocities;
    local_params.forces = forces;
    local_params.kinetic = kinetic;
    local_params.potential = potential;
    local_params.neighbors = neighbors;
    local_params.counts = counts;
    hipMemcpyToSymbol(HIP_SYMBOL(params), &local_params, sizeof(GlobalConstants));

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "INIT: A CUDA error occured: code=%d, %s, %s\n", errCode, hipGetErrorName(errCode), hipGetErrorString(errCode));
        exit(-1);
    }

    int threadsPerBlock = 512;
    int blocks = (numberOfParticles * numberOfParticles + threadsPerBlock - 1) / threadsPerBlock;
    hipMemset(forces, 0, 3 * numberOfParticles * sizeof(float));
    calculateForceKernel<<<blocks, threadsPerBlock>>>();
    hipDeviceSynchronize();
    blocks = (numberOfParticles * 2 + threadsPerBlock - 1) / threadsPerBlock;
    calculateBondAngleKernel<<<blocks, threadsPerBlock>>>();
    hipDeviceSynchronize();

    errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "FORCE1: A CUDA error occured: code=%d, %s, %s\n", errCode, hipGetErrorName(errCode), hipGetErrorString(errCode));
        exit(-1);
    }

    timesteps = 0;
}

CudaSim::~CudaSim()
{
    hipFree(positions);
    hipFree(velocities);
    hipFree(forces);
    hipFree(neighbors);
    hipFree(counts);
    hipFree(potential);
    hipFree(kinetic);
}

void CudaSim::advance()
{
    // size_t startTime;
    // size_t endTime;

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "ADVANCE: A CUDA error occured: code=%d, %s, %s\n", errCode, hipGetErrorName(errCode), hipGetErrorString(errCode));
        exit(-1);
    }

    int threadsPerBlock = 512;
    int blocks = (numberOfParticles * 3 + threadsPerBlock - 1) / threadsPerBlock;
    hipMemset(kinetic, 0, sizeof(float));
    // startTime = CycleTimer::currentTicks();
    calculateKineticKernel<<<blocks, threadsPerBlock>>>();
    hipDeviceSynchronize();
    // endTime = CycleTimer::currentTicks();
    // printf("Kinetic Kernel took %lu seconds\n", endTime - startTime);


    errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "KINETIC: A CUDA error occured: code=%d, %s, %s\n", errCode, hipGetErrorName(errCode), hipGetErrorString(errCode));
        exit(-1);
    }

    // startTime = CycleTimer::currentTicks();
    updateVelocityKernel<<<blocks, threadsPerBlock>>>();
    hipDeviceSynchronize();
    // endTime = CycleTimer::currentTicks();
    // printf("Velocity Kernel took %lu seconds\n", endTime - startTime);

    errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "VELOCITY: A CUDA error occured: code=%d, %s, %s\n", errCode, hipGetErrorName(errCode), hipGetErrorString(errCode));
        exit(-1);
    }

    // startTime = CycleTimer::currentTicks();
    updatePositionKernel<<<blocks, threadsPerBlock>>>();
    hipDeviceSynchronize();
    // endTime = CycleTimer::currentTicks();
    // printf("Position Kernel took %lu seconds\n", endTime - startTime);

    errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "POSITION: A CUDA error occured: code=%d, %s, %s\n", errCode, hipGetErrorName(errCode), hipGetErrorString(errCode));
        exit(-1);
    }

    // Every 50 timesteps, update the neighbors list
    if (timesteps % 50 == 0)
    {
        // Mark neighbors
        blocks = (numberOfParticles * numberOfParticles + threadsPerBlock - 1) / threadsPerBlock;
        markNeighbors<<<blocks, threadsPerBlock>>>();

        // Scan
        thrust::device_ptr<short> th_neighbors = thrust::device_pointer_cast(neighbors);
        thrust::exclusive_scan(th_neighbors, th_neighbors + (numberOfParticles * numberOfParticles), th_neighbors);

        blocks = (numberOfParticles + threadsPerBlock - 1) / threadsPerBlock;
        getNeighborCounts<<<blocks, threadsPerBlock>>>();

        // Reduce
        reduceNeighbors<<<blocks, threadsPerBlock>>>();
    }
    timesteps += 1;
    // startTime = CycleTimer::currentTicks();
    // printf("Launching %lu threads\n", numberOfParticles * numberOfParticles);
    blocks = (numberOfParticles * numberOfParticles / 5 + threadsPerBlock - 1) / threadsPerBlock;
    hipMemset(forces, 0, 3 * numberOfParticles * sizeof(float));

    calculateForceKernel<<<blocks, threadsPerBlock>>>();
    hipDeviceSynchronize();
    // endTime = CycleTimer::currentTicks();
    // printf("CalculateForce Kernel took %lu seconds\n", endTime - startTime);
    // startTime = CycleTimer::currentTicks();
    blocks = (numberOfParticles * 3 + threadsPerBlock - 1) / threadsPerBlock;
    calculateBondAngleKernel<<<blocks, threadsPerBlock>>>();
    hipDeviceSynchronize();
    // endTime = CycleTimer::currentTicks();
    // printf("BondAndAngle Kernel took %lu seconds\n", endTime - startTime);

    errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "FORCE2: A CUDA error occured: code=%d, %s, %s\n", errCode, hipGetErrorName(errCode), hipGetErrorString(errCode));
        exit(-1);
    }

    // startTime = CycleTimer::currentTicks();
    blocks = (numberOfParticles * 3 + threadsPerBlock - 1) / threadsPerBlock;
    updateVelocityKernel<<<blocks, threadsPerBlock>>>();
    hipDeviceSynchronize();
    // endTime = CycleTimer::currentTicks();
    // printf("Velocity Kernel took %lu seconds\n", endTime - startTime);

    errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "VELOCITY: A CUDA error occured: code=%d, %s, %s\n", errCode, hipGetErrorName(errCode), hipGetErrorString(errCode));
        exit(-1);
    }

    // startTime = CycleTimer::currentTicks();
    calculateKineticKernel<<<blocks, threadsPerBlock>>>();
    hipDeviceSynchronize();
    // endTime = CycleTimer::currentTicks();
    // printf("Velocity Kernel took %lu seconds\n", endTime - startTime);

    errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "KINETIC: A CUDA error occured: code=%d, %s, %s\n", errCode, hipGetErrorName(errCode), hipGetErrorString(errCode));
        exit(-1);
    }

}