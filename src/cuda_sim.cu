#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_runtime.h>

#include "cuda_sim.h"

struct GlobalConstants {
    size_t numberOfParticles;

    float boxSize;

    float dudr;
    float r_cut;
    float u_cut;
    float delta;

    float* positions;
    float* velocities;
    float* forces;

    float* kinetic;
    float* potential;
};
__constant__ GlobalConstants params;

/* Device functions */

__device__ float calculateDistance(size_t i, size_t j)
{
    float boxSize = params.boxSize;

    float distance = 0;
    for (int k = 0; k < 3; k++){
        float diff = params.positions[i * 3 + k] - params.positions[j * 3 + k];
        if (diff < -boxSize/2) diff += boxSize;
        if (diff > boxSize/2) diff -= boxSize;
        distance += diff * diff;
    }

    distance = std::sqrt(distance);

    return distance;
}

/* Kernels */

__global__ void calculateForceKernel()
{
    int numberOfParticles = params.numberOfParticles;
    float boxSize = params.boxSize;
    float dudr = params.dudr;
    float r_cut = params.r_cut;
    float u_cut = params.u_cut;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int i = index % numberOfParticles;
    int j = index / numberOfParticles;

    if (i >= numberOfParticles || j >= numberOfParticles) return;
    if (j <= i) return;
    if ((i % 3 == 0) && (j - i == 1) || (i % 3 == 0) && (j - i == 2) ) return;

    float dist = calculateDistance(i, j);
    if (dist >= r_cut) return;

    float u_actual = 4 * (1/std::pow(dist, 12) - 1/std::pow(dist, 6));
    atomicAdd(params.potential, u_actual - u_cut - (dist - r_cut) * dudr);

    for (int k = 0; k < 3; k++)
    {
        float r = params.positions[i * 3 + k] - params.positions[j * 3 + k];
        if (r < -boxSize/2) r += boxSize;
        if (r > boxSize/2) r -= boxSize;

        float f = r * (48/pow(dist, 14) - 24/pow(dist, 8) + dudr/dist);
        atomicAdd(&params.forces[i * 3 + k], f);
        atomicAdd(&params.forces[j * 3 + k], -1 * f);
    }

}

__global__ void calculateBondAngleKernel()
{
    int numberOfParticles = params.numberOfParticles;
    float boxSize = params.boxSize;
    double spring = 1000;
    double x0 = 0.8;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= numberOfParticles * 2) return;
    int h = index % 2 + 1;
    int i = (index / 2) * 3;

    float dist = calculateDistance(i, i + h);

    for (int k = 0; k < 3; k++){
        double r = params.positions[i * 3 + k] - params.positions[(i+h) * 3 + k];
        if (r < -boxSize/2) r += boxSize;
        if (r > boxSize/2) r -= boxSize;
        atomicAdd(params.potential, 0.5 * spring * pow(dist - x0, 2));
        float f = -spring * (dist - x0) * r / dist;
        atomicAdd(&params.forces[i * 3 + k], f);
        atomicAdd(&params.forces[(i + h) * 3 + k], -f);
    }
}

__global__ void calculateKineticKernel()
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index > params.numberOfParticles * 3) return;
    int i = index / 3;
    int k = index % 3;

    atomicAdd(params.kinetic, 0.5 * pow(params.velocities[i * 3 + k], 2));
}

__global__ void updatePositionKernel()
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index > params.numberOfParticles * 3) return;
    int i = index / 3;
    int k = index % 3;

    params.positions[i * 3 + k] += params.velocities[i * 3 + k] * params.delta;
    if (params.positions[i * 3 + k] > params.boxSize){
        params.positions[i * 3 + k] -= params.boxSize;
    }
    if (params.positions[i * 3 + k] < 0){
        params.positions[i * 3 + k] += params.boxSize;
    }
}

__global__ void updateVelocityKernel()
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index > params.numberOfParticles * 3) return;
    int i = index / 3;
    int k = index % 3;

    float a = params.forces[i * 3 + k] * params.delta / 2;
    params.velocities[i * 3 + k] += a;
}

/* Interface */

void CudaSim::getPositions(float *out)
{
    hipMemcpy(out, positions, 3 * numberOfParticles * sizeof(float), hipMemcpyDeviceToHost);
}

void CudaSim::getVelocities(float *out)
{
    hipMemcpy(out, velocities, 3 * numberOfParticles * sizeof(float), hipMemcpyDeviceToHost);
}

float CudaSim::getKinetic()
{
    float out;
    hipMemcpy(&out, kinetic, sizeof(float), hipMemcpyDeviceToHost);
    return out;
}

float CudaSim::getPotential()
{
    float out;
    hipMemcpy(&out, potential, sizeof(float), hipMemcpyDeviceToHost);
    return out;
}

void printDeviceInfo()
{
    int deviceCount = 0;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}

CudaSim::CudaSim(size_t numberOfParticles_in, float boxSize_in, float dudr_in,
                 float r_cut_in, float u_cut_in, float delta_in, float* positions_init)
{
    printDeviceInfo();

    // Initialize instance variables
    numberOfParticles = numberOfParticles_in;
    boxSize = boxSize_in;
    dudr = dudr_in;
    r_cut = r_cut_in;
    u_cut = u_cut_in;
    delta = delta_in;
    hipMalloc(&positions, sizeof(float) * 3 * numberOfParticles);
    hipMemcpy(positions, positions_init, sizeof(float) * 3 * numberOfParticles, hipMemcpyHostToDevice);
    hipMalloc(&velocities, sizeof(float) * 3 * numberOfParticles);
    hipMemset(velocities, 0, sizeof(float) * 3 * numberOfParticles);
    hipMalloc(&forces, sizeof(float) * 3 * numberOfParticles);
    hipMemset(forces, 0, sizeof(float) * 3 * numberOfParticles);
    hipMalloc(&kinetic, sizeof(float));
    hipMemset(kinetic, 0, sizeof(float));
    hipMalloc(&potential, sizeof(float));
    hipMemset(potential, 0, sizeof(float));

    // Initialize global constants
    GlobalConstants local_params;
    local_params.boxSize = boxSize;
    local_params.dudr = dudr;
    local_params.r_cut = r_cut;
    local_params.u_cut = u_cut;
    local_params.delta = delta;
    local_params.numberOfParticles = numberOfParticles;
    local_params.positions = positions;
    local_params.velocities = velocities;
    local_params.forces = forces;
    local_params.kinetic = kinetic;
    local_params.potential = potential;
    hipMemcpyToSymbol(HIP_SYMBOL(params), &local_params, sizeof(GlobalConstants));

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "INIT: A CUDA error occured: code=%d, %s, %s\n", errCode, hipGetErrorName(errCode), hipGetErrorString(errCode));
        exit(-1);
    }

    int threadsPerBlock = 512;
    int blocks = (numberOfParticles * numberOfParticles + threadsPerBlock - 1) / threadsPerBlock;
    hipMemset(forces, 0, 3 * numberOfParticles * sizeof(float));
    calculateForceKernel<<<blocks, threadsPerBlock>>>();
    hipDeviceSynchronize();
    blocks = (numberOfParticles * 2 + threadsPerBlock - 1) / threadsPerBlock;
    calculateBondAngleKernel<<<blocks, threadsPerBlock>>>();
    hipDeviceSynchronize();

    errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "FORCE1: A CUDA error occured: code=%d, %s, %s\n", errCode, hipGetErrorName(errCode), hipGetErrorString(errCode));
        exit(-1);
    }
}

CudaSim::~CudaSim()
{
    hipFree(positions);
    hipFree(velocities);
    hipFree(forces);
}

void CudaSim::advance()
{

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "ADVANCE: A CUDA error occured: code=%d, %s, %s\n", errCode, hipGetErrorName(errCode), hipGetErrorString(errCode));
        exit(-1);
    }

    int threadsPerBlock = 512;
    int blocks = (numberOfParticles * 3 + threadsPerBlock - 1) / threadsPerBlock;
    hipMemset(kinetic, 0, sizeof(float));
    calculateKineticKernel<<<blocks, threadsPerBlock>>>();
    hipDeviceSynchronize();

    errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "KINETIC: A CUDA error occured: code=%d, %s, %s\n", errCode, hipGetErrorName(errCode), hipGetErrorString(errCode));
        exit(-1);
    }

    updateVelocityKernel<<<blocks, threadsPerBlock>>>();
    hipDeviceSynchronize();

    errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "VELOCITY: A CUDA error occured: code=%d, %s, %s\n", errCode, hipGetErrorName(errCode), hipGetErrorString(errCode));
        exit(-1);
    }

    updatePositionKernel<<<blocks, threadsPerBlock>>>();
    hipDeviceSynchronize();

    errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "POSITION: A CUDA error occured: code=%d, %s, %s\n", errCode, hipGetErrorName(errCode), hipGetErrorString(errCode));
        exit(-1);
    }

    blocks = (numberOfParticles * numberOfParticles + threadsPerBlock - 1) / threadsPerBlock;
    hipMemset(forces, 0, 3 * numberOfParticles * sizeof(float));
    calculateForceKernel<<<blocks, threadsPerBlock>>>();
    hipDeviceSynchronize();
    blocks = (numberOfParticles * 3 + threadsPerBlock - 1) / threadsPerBlock;
    calculateBondAngleKernel<<<blocks, threadsPerBlock>>>();
    hipDeviceSynchronize();

    errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "FORCE2: A CUDA error occured: code=%d, %s, %s\n", errCode, hipGetErrorName(errCode), hipGetErrorString(errCode));
        exit(-1);
    }

    blocks = (numberOfParticles * 3 + threadsPerBlock - 1) / threadsPerBlock;
    updateVelocityKernel<<<blocks, threadsPerBlock>>>();
    hipDeviceSynchronize();

    errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "VELOCITY: A CUDA error occured: code=%d, %s, %s\n", errCode, hipGetErrorName(errCode), hipGetErrorString(errCode));
        exit(-1);
    }

    calculateKineticKernel<<<blocks, threadsPerBlock>>>();
    hipDeviceSynchronize();

    errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "KINETIC: A CUDA error occured: code=%d, %s, %s\n", errCode, hipGetErrorName(errCode), hipGetErrorString(errCode));
        exit(-1);
    }

}