#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "particle.h"
#include "md.h"

struct GlobalConstants {
    size_t numberOfParticles;

    float boxSize;

    double dudr;
    double r_cut;
    double u_cut;
    double delta;
};
__constant__ GlobalConstants cuConstParams;

__device__ float* forces;
__device__ float* positions;
__device__ float* velocities;
__device__ double kinetic;
__device__ double potential;

size_t numberOfParticles;

__device__ double calculateDistance_c(size_t i, size_t j)
{
    float boxSize = cuConstParams.boxSize;

    double distance = 0;
    for (int k = 0; k < 3; k++){
        double diff = positions[i * 3 + k] - positions[j * 3 + k];
        if (diff < -boxSize/2) diff += boxSize;
        if (diff > boxSize/2) diff -= boxSize;
        distance += diff * diff;
    }

    distance = std::sqrt(distance);
    return distance;
}

__global__ void calculateForceKernel()
{
    int numberOfParticles = cuConstParams.numberOfParticles;
    float boxSize = cuConstParams.boxSize;
    double dudr = cuConstParams.dudr;
    double r_cut = cuConstParams.r_cut;
    double u_cut = cuConstParams.u_cut;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int i = index % numberOfParticles;
    int j = index / numberOfParticles;

    if ((i % 3 == 0) && (j - i == 1) || (i % 3 == 0) && (j - i == 2) ) return;

    double dist = calculateDistance_c(i, j);
    if (dist >= r_cut) return;

    double u_actual = 4 * (1/std::pow(dist, 12) - 1/std::pow(dist, 6));
    potential += u_actual - u_cut - (dist - r_cut) * dudr; // TODO: Atomic addd

    for (int k = 0; k < 3; k++)
    {
        double r = positions[i * 3 + k] - positions[j * 3 + k];
        if (r < -boxSize/2) r += boxSize;
        if (r > boxSize/2) r -= boxSize;

        double f = r * (48/pow(dist, 14) - 24/pow(dist, 8) + dudr/dist);
        forces[i * numberOfParticles + k] += f;
        forces[j * numberOfParticles + k] -= f;
    }
}

void calculateForceAndEnergyCuda()
{
    hipDeviceSynchronize();
    const int threadsPerBlock = 512;
    const int blocks = (numberOfParticles + threadsPerBlock - 1) / threadsPerBlock;

    calculateForceKernel<<<blocks, threadsPerBlock>>>();
}

__global__ void calculateKineticKernel()
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index > cuConstParams.numberOfParticles * 3) return;
    int i = index % cuConstParams.numberOfParticles;
    int k = index / 3;

    kinetic += 0.5 * pow(velocities[i * 3 + k], 2);
}

void calculateKineticCuda()
{
    hipDeviceSynchronize();
    const int threadsPerBlock = 512;
    const int blocks = (numberOfParticles + threadsPerBlock - 1) / threadsPerBlock;

    double* energy_v;
    hipGetSymbolAddress((void **)&energy_v, "kinetic");
    hipMemset(energy_v, 0, sizeof(double));

    calculateKineticKernel<<<blocks, threadsPerBlock>>>();
}

__global__ void updatePositionKernel()
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index > cuConstParams.numberOfParticles * 3) return;
    int i = index % cuConstParams.numberOfParticles;
    int k = index / 3;

    positions[i * 3 + k] += velocities[i * 3 + k] * cuConstParams.delta;
    if (positions[i * 3 + k] > cuConstParams.boxSize){
        positions[i * 3 + k] -= cuConstParams.boxSize;
    }
    if (positions[i * 3 + k] < 0){
        positions[i * 3 + k] += cuConstParams.boxSize;
    }
}

void updatePositionCuda()
{
    hipDeviceSynchronize();
    const int threadsPerBlock = 512;
    const int blocks = (numberOfParticles + threadsPerBlock - 1) / threadsPerBlock;

    updatePositionKernel<<<blocks, threadsPerBlock>>>();
}

__global__ void updateVelocityKernel()
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index > cuConstParams.numberOfParticles * 3) return;
    int i = index % cuConstParams.numberOfParticles;
    int k = index / 3;


    double a = forces[i * 3 + k] * cuConstParams.delta;
    velocities[i * 3 + k] += a;
}

void updateVelocityCuda()
{
    hipDeviceSynchronize();
    const int threadsPerBlock = 512;
    const int blocks = (numberOfParticles + threadsPerBlock - 1) / threadsPerBlock;

    updateVelocityKernel<<<blocks, threadsPerBlock>>>();
}

void getPositions(float *out)
{
    printf("Getting positions from CUDA device\n");
    // Copies positions into parameter from device
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(out, HIP_SYMBOL(positions), 3 * numberOfParticles * sizeof(float));
}

void getKinetic(double *out)
{
    // Copies kinetic energy from device to out
    hipMemcpyFromSymbol(out, HIP_SYMBOL(kinetic), sizeof(float));
}

void getPotential(double *out)
{
    // Copies potential energy from device to out
    hipMemcpyFromSymbol(out, HIP_SYMBOL(potential), sizeof(float));
}

void initializeCuda(size_t numberOfParticles_in, float boxSize, double dudr,
                double r_cut, double u_cut, double delta, float* init_positions)
{
    printf("Initializing CUDA\n");
    // Initialize Global constant parameters
    GlobalConstants params;
    params.boxSize = boxSize;
    params.boxSize = dudr;
    params.boxSize = boxSize;
    params.r_cut = r_cut;
    params.u_cut = u_cut;
    params.delta = delta;
    params.numberOfParticles = numberOfParticles_in;
    numberOfParticles = numberOfParticles_in;
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstParams), &params, sizeof(GlobalConstants));

    // Initialize kinetic and potential
    double* energy_v;
    hipGetSymbolAddress((void **)&energy_v, "kinetic");
    hipMemset(energy_v, 0, sizeof(double));
    hipGetSymbolAddress((void **)&energy_v, "potential");
    hipMemset(energy_v, 0, sizeof(double));

    // Initialize positions, forces, and velocities
    hipMemcpyToSymbol(HIP_SYMBOL(positions), &init_positions, 3 * numberOfParticles * sizeof(float));
    float* device_v;
    hipGetSymbolAddress((void **)&device_v, "forces");
    hipMemset(device_v, 0, 3 * numberOfParticles * sizeof(double));
    hipGetSymbolAddress((void **)&device_v, "velocities");
    hipMemset(device_v, 0, 3 * numberOfParticles * sizeof(double));
}